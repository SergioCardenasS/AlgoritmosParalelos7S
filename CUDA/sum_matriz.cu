#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

__global__
void sum_mat(float* A,float* B,float* C,const int N,const int M)
{
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < M*N)
        C[id] = A[id] + B[id];
}

__global__
void sum_mat_r(float* A,float* B,float* C,int N,int M)
{
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if(id<N)
    {
        int i;
        for(i = 0;i < M; ++i)
            C[M*id+i]=A[M*id+i]+B[M*id+i];
    }
}

__global__
void sum_mat_c(float* A,float* B,float* C,int N,int M)
{
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < M)
    {
        int i;
        for(i=0;i<N;++i)
            C[M*i+id]=A[M*i+id]+B[M*i+id];
    }
}

// debuging functions
void init_array(float *a, const int N);
void init_mat(float *a, const int N, const int M);
void init_mat_c(float *a, const int N, const int M);
void print_array(float *a, const int N, char *d);
void print_mat(float *a, const int N, const int M, char *d);

int main (void) 
{
    srand( time(NULL) );

    float *a, *b, *c;
    float *dev_a, *dev_b, *dev_c;
    int N=4;
    int M=4;
    int size=N*M;
    int device_size=size*sizeof(float);
    a=(float*)malloc(device_size);
    b=(float*)malloc(device_size);
    c=(float*)malloc(device_size);
    init_mat(a, N, M);
    init_mat(b, N, M);
    init_mat_c(c, N, M);

    printf("<<<<<<<<<< initial data:\n");
    
    print_mat(a, N, M, "matrix A ");
    print_mat(b, N, M, "matrix B ");

    hipMalloc((void**)&dev_a, device_size);
    hipMalloc((void**)&dev_b, device_size);
    hipMalloc((void**)&dev_c, device_size);

    hipMemcpy(dev_a, a, device_size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, device_size, hipMemcpyHostToDevice);

    printf("\n\nRunning Kernel...\n\n");
    //sum_mat<<<N*M,M>>>(dev_a, dev_b, dev_c, N, M);
    sum_mat_c<<<N,N>>>(dev_a, dev_b, dev_c, N, M);
    //sum_mat_r<<<N*M,M>>>(dev_a, dev_b, dev_c, N, M);
    //printf("error code: %s\n",cudaGetErrorString(cudaGetLastError()));

    hipMemcpy(c, dev_c, device_size, hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    printf(">>>>>>>>>> final data:\n");
    print_mat(c,N, M, "Matriz final");
    return 0;
};

void init_mat(float *a, const int N, const int M) 
{
    int i, j;
    for(i=0; i<N; i++)
        for(j=0; j<M; j++)
            a[i*M+j] = rand() % 4 + 1;
}

void init_mat_c(float *a, const int N, const int M) 
{
    int i, j;
    for(i=0; i<N; i++)
        for(j=0; j<M; j++)
            a[i*M+j] = 0;
}
void print_mat(float *a, const int N, const int M, char *d) 
{
    int i, j;
    for(i=0; i<N; i++)
    {
	    printf("\n%s[%d]:", d, i);
    	for (j=0; j<M; j++)
            printf("\t%6.4f", a[i*M+j]);
    }
    printf("\n");
}